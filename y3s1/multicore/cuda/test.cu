// File : test.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
    printf("cuda capable\n");
}

int main()
{
    hello<<<1,10>>>();
    hipDeviceSynchronize();
    return 1;
}